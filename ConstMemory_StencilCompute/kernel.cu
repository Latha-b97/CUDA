#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

//My Notes : Refer Udemy Course - Const memory
// 1. Const memory is a special purpose memory used for data that is read only from device and accessed by all threads in the warp
// 2. Constant memory is read-only from kernel, but both readable and writable from host.
// 3. Constant memeory works best if all threads access the same location in the constant memory.
// 4. Constant memory variables exist for the lifespan of the application that are accessible from all threads with in the grid and by the host 

#define c0 1
#define c1 2
#define c2 3
#define c3 4
#define c4 5

#define RADIUS 4

#define BLOCKDIM 128

//constant memory declaration
__constant__ int coef[9];

void initialize(int* input, const int array_size)
{
	// fill with 1 to 10
	for (int i = 0; i < array_size; i++)
	{
		input[i] = i % 10;
	}
}

void compare_arrays(int* a, int* b, int size)
{
	for (int i = 0; i < size; i++)
	{
		if (a[i] != b[i])
		{
			printf("Arrays are different \n");
			printf("%d - %d | %d \n", i, a[i], b[i]);
			//return;
		}
	}
	printf("Arrays are same \n");
}

// stencil calculation in host side.
// This is for verification purpose to compare with Device values
void host_const_calculation(int* in, int* out, int size)
{
	for (int i = 0; i < size; i++)
	{

		if (i < RADIUS)
		{
			out[i] = in[i + 4] * c0
				+ in[i + 3] * c1
				+ in[i + 2] * c2
				+ in[i + 1] * c3
				+ in[i] * c4;

			if (i == 3)
			{
				out[i] += in[2] * c3;
				out[i] += in[1] * c2;
				out[i] += in[0] * c1;
			}
			else if (i == 2)
			{
				out[i] += in[1] * c3;
				out[i] += in[0] * c2;
			}
			else if (i == 1)
			{
				out[i] += in[0] * c3;
			}
		}
		else if ((i + RADIUS) >= size)
		{
			out[i] = in[i - 4] * c0
				+ in[i - 3] * c1
				+ in[i - 2] * c2
				+ in[i - 1] * c3
				+ in[i] * c4;

			if (i == size - 4)
			{
				out[i] += in[size - 3] * c3;
				out[i] += in[size - 2] * c2;
				out[i] += in[size - 1] * c1;
			}
			else if (i == size - 3)
			{
				out[i] += in[size - 2] * c3;
				out[i] += in[size - 1] * c2;
			}
			else if (i == size - 2)
			{
				out[i] += in[size - 1] * c3;
			}
		}
		else
		{
			out[i] = (in[i - 4] + in[i + 4]) * c0
				+ (in[i - 3] + in[i + 3]) * c1
				+ (in[i - 2] + in[i + 2]) * c2
				+ (in[i - 1] + in[i + 1]) * c3
				+ in[i] * c4;
		}
	}
}

//setting up constant memory from host
// write the coofficient to the constant memory and pass it to the device memory.
void setup_coef_1()
{
	const int h_coef[] = { c0,c1,c2,c3,c4,c3,c2,c1,c0 };

	// this transfers constant memory to device using hipMemcpyToSymbol function
	hipMemcpyToSymbol(HIP_SYMBOL(coef), h_coef, (9) * sizeof(float));
}

// Here is the kernel with one dimensional threadblock with multiple threads
__global__ void constant_stencil_smem_test(int* in, int* out, int size)
{
	//Here we are declaring shared memory.We need radius amount of elements from previous and next data blocks. 
	//We are padding our shared memory with Radius amount at the beginning and end
	// These padded memory location will store the previous and next thread block
	__shared__ int smem[BLOCKDIM + 2 * RADIUS];

	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	//We need to consider Halo elements if the element that we need to compute are in the sides.
	int bid = blockIdx.x;  // this is first thread block have threadId as 0
	int num_of_blocks = gridDim.x; // this is the last thread block which has blockIdx.x as size-1


	int value = 0;

	if (gid < size)
	{
		//index with offset
		int sidx = threadIdx.x + RADIUS;

		//load data to shared mem
		smem[sidx] = in[gid];

		// load elements from previous and next data block part

		// This is for the middle thread block, noneed to consider Halo elements 
		if (bid != 0 && bid != (num_of_blocks - 1))
		{
			if (threadIdx.x < RADIUS)
			{
				smem[sidx - RADIUS] = in[gid - RADIUS];
				smem[sidx + BLOCKDIM] = in[gid + BLOCKDIM];
			}
		}
		// For the first thread block, we need to consider Halo elements in the beginning
		else if (bid == 0)
		{
			if (threadIdx.x < RADIUS)
			{
				smem[sidx - RADIUS] = 0;
				smem[sidx + BLOCKDIM] = in[gid + BLOCKDIM];
			}
		}
		// For the last thread block, we need to consider Halo elements in the end
		else
		{
			if (threadIdx.x < RADIUS)
			{
				smem[sidx - RADIUS] = in[gid - RADIUS];
				smem[sidx + BLOCKDIM] = 0;
			}
		}

		// So far, we have successfully stored the elements in the shared memory
		// then we need to wait untill all the threads in block finish storing smem
		__syncthreads();

		value += smem[sidx - 4] * coef[0];
		value += smem[sidx - 3] * coef[1];
		value += smem[sidx - 2] * coef[2];
		value += smem[sidx - 1] * coef[3];
		value += smem[sidx - 0] * coef[4];
		value += smem[sidx + 1] * coef[5];
		value += smem[sidx + 2] * coef[6];
		value += smem[sidx + 3] * coef[7];
		value += smem[sidx + 4] * coef[8];

		// finally store the final result to output array
		out[gid] = value;
	}
}

int main(int argc, char ** argv)
{
	int size = 1 << 22;
	int byte_size = sizeof(int) * size;
	int block_size = BLOCKDIM;

	int * h_in, *h_out, *h_ref;

	h_in = (int*)malloc(byte_size);
	h_out = (int*)malloc(byte_size);
	h_ref = (int*)malloc(byte_size);

	initialize(h_in, size); // fill with 1 to 10

	int * d_in, *d_out;
	hipMalloc((void**)&d_in, byte_size);
	hipMalloc((void**)&d_out, byte_size);

	hipMemcpy(d_in, h_in, byte_size, hipMemcpyHostToDevice);
	hipMemset(d_out, 0, byte_size);

	setup_coef_1();

	dim3 blocks(block_size);
	dim3 grid(size / blocks.x);

	constant_stencil_smem_test << < grid, blocks >> > (d_in, d_out, size);
	hipDeviceSynchronize();

	hipMemcpy(h_ref, d_out, byte_size, hipMemcpyDeviceToHost);

	host_const_calculation(h_in, h_out, size);

	printf("Comparing CPU and GPU results \n");
	compare_arrays(h_ref, h_out, size);

	hipFree(d_out);
	hipFree(d_in);
	free(h_ref);
	free(h_out);
	free(h_in);

	return 0;
}